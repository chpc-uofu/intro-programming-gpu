#include "hip/hip_runtime.h"
#include <mul.h>

__global__ void MatrixMulKernel1(double *M_d, double *N_d, 
                                 double *P_d, int const SZ)
{
    double Pval=0;
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    for(int k=0; k<SZ; k++)
        Pval+=M_d[tx*SZ +k]*N_d[k*SZ+ty];
    P_d[tx*SZ+ty]=Pval;    
    return;
}

