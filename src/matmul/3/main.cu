#include "aux.h"
#include "mul.h"

int main(void)
{
    double *M_h, *N_h, *P_h;  // Pointers (host)
    double *M_d, *N_d, *P_d;  // Pointers (device)
    int const SZ=512;
    int const SZ2=SZ*SZ;

    // Allocate the matrices M, N & P on the host
    M_h=allocVectorOnHost(SZ2);
    N_h=allocVectorOnHost(SZ2);
    P_h=allocVectorOnHost(SZ2);

    // Initialize the matrices M, N (host)
    for(int k=0; k<SZ2; k++)
        M_h[k]=(double)k;
    for(int k=0; k<SZ2; k++)
        N_h[k]=k+1.;

    // Allocate M, N and P on the device
    if(hipMalloc(&M_d,sizeof(double)*SZ2) != hipSuccess){
       printf(" ERROR: alloc vector M on DEVICE\n");
       return 1;}

    if(hipMalloc(&N_d,sizeof(double)*SZ2) != hipSuccess){
       printf(" ERROR: alloc vector N on DEVICE\n");
       return 1;}
    
    if(hipMalloc(&P_d,sizeof(double)*SZ2) != hipSuccess){
       printf(" ERROR: alloc vector P on DEVICE \n");
       return 1;}


    // Copy M,N from Host onto Device
    if(hipMemcpy(M_d,M_h,sizeof(double)*SZ2,hipMemcpyHostToDevice) != hipSuccess){
       printf(" ERROR: copy vector M: HOST -> DEVICE\n");
       return 1;}
    
    if(hipMemcpy(N_d,N_h,sizeof(double)*SZ2,hipMemcpyHostToDevice) != hipSuccess){
       printf(" ERROR: copy vector N: HOST -> DEVICE\n");
       return 1;}


    // KERNEL: P=M*N on DEVICE 
    int const THREADX=16;
    int const THREADY=16;
    dim3 dimBlock(THREADX,THREADY,1);
    int numBlocksX=(SZ%THREADX==0 ? SZ/THREADX : SZ/THREADX +1);
    int numBlocksY=(SZ%THREADY==0 ? SZ/THREADY : SZ/THREADY +1);
    dim3 dimGrid(numBlocksX,numBlocksY,1);

    printf(" Calling Kernel ...\n");
    MatrixMulKernel3<<<dimGrid,dimBlock>>>(M_d,N_d,P_d,SZ);
    printf(" Kernel Call Finished ...\n");

    if(hipSuccess != hipGetLastError()){
       printf(" ERROR: MatrixMulKernel\n");
       return 1;}


    // Copy P (result) from the Device to the Host
    if(hipMemcpy(P_h,P_d,sizeof(double)*SZ2,hipMemcpyDeviceToHost) != hipSuccess){
       printf(" ERROR: copy vector N: DEVICE -> HOST\n");
       return 1;}


    #ifdef check
       // Calc. P=M*N on the Host
       double *P = matrixMulHost(M_h,N_h,SZ);

       // Calc. Frob. Norm of (P-P_h)
       double frobnorm = calcDiff(P,P_h,SZ);
       printf("\n Frob. Norm(P-P_h):%16.10lf\n\n", frobnorm);

       // Deallocate P
       free(P);
    #endif


    // Deallocate matrices on the Host (M_h, N_h, P_h)
    free(M_h);
    free(N_h);
    free(P_h);


    // Deallocate matrices on the Device (M_d, N_d, P_d)
    if(hipFree(M_d) != hipSuccess){
       printf(" ERROR: unable to deallocate M_d (DEVICE)\n");
       return 1;}

    if(hipFree(N_d) != hipSuccess){
       printf(" ERROR: unable to deallocate N_d (DEVICE)\n");
       return 1;}

    if(hipFree(P_d) != hipSuccess){
       printf(" ERROR: unable to deallocate P_d (DEVICE)\n");
       return 1;}

    return 0;
}
