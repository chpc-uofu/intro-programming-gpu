// Copy M from Host onto Device
if(hipMemcpy(M_d, M_h, sizeof(double)*SZ2,
	      hipMemcpyHostToDevice) != hipSuccess)
{
   printf(" ERROR: copy vector M: HOST -> DEVICE\n");
   return 1;
}


// Copy P (result) from the Device to the Host
if(hipMemcpy(P_h, P_d, sizeof(double)*SZ2,
	      hipMemcpyDeviceToHost) != hipSuccess)
{
   printf(" ERROR: copy vector N: DEVICE -> HOST\n");
   return 1;
}
